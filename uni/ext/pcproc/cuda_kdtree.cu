#include "hip/hip_runtime.h"
#include "cuda_kdtree.cuh"
// #define THRUST_DEBUG 1
#include <hip/hip_runtime.h>
#include <thrust/gather.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <hip/hip_vector_types.h>
#include "cutil_math.h"
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <thrust/count.h>
#include <hip/hip_vector_types.h>

#include <limits>

__device__ __forceinline__
float infinity()
{
    return __int_as_float(0x7f800000);
}

#ifndef INFINITY
#define INFINITY infinity()
#endif

namespace tinyflann
{
    namespace cuda {

        template <class T>
        __device__ __host__ void swap( T& x, T& y ) {
            T t=x; x=y; y=t;
        }

        template <class GreaterThan, class RandomAccessIterator, class RandomAccessIterator2>
        __host__ __device__ void
        sift_down( RandomAccessIterator key, RandomAccessIterator2 value, size_t begin, size_t length, GreaterThan c = GreaterThan() )
        {

            while( 2*begin+1 < length ) {
                size_t left = 2*begin+1;
                size_t right = 2*begin+2;
                size_t largest=begin;
                if((left < length)&& c(key[left], key[largest]) ) largest=left;

                if((right < length)&& c(key[right], key[largest]) ) largest=right;

                if( largest != begin ) {
                    cuda::swap( key[begin], key[largest] );
                    cuda::swap( value[begin], value[largest] );
                    begin=largest;
                }
                else return;
            }
        }

//! creates a max-heap in the array beginning at begin of length "length"
//! if c is a less-than comparator, it will create a min-heap
        template <class GreaterThan, class RandomAccessIterator, class RandomAccessIterator2>
        __host__ __device__ void
        make_heap( RandomAccessIterator key,  RandomAccessIterator2 value, size_t length, GreaterThan c = GreaterThan() )
        {
            int i=length/2-1;
            while( i>=0 ) {
                sift_down( key, value, i, length, c );
                i--;
            }
        }

        template< typename DistanceType >
        struct GreaterThan
        {
            __device__
            bool operator()(DistanceType a, DistanceType b)
            {
                return a>b;
            }
        };

//! result set for the 1nn search. Doesn't do any global memory accesses on its own,
        template<typename DistanceType>
        struct SingleResultSet {
            int bestIndex;
            DistanceType bestDist;
            const DistanceType epsError;

            __device__ __host__ SingleResultSet(DistanceType eps) : bestIndex(-1), bestDist(INFINITY), epsError(eps) {}

            __device__
            inline float
            worstDist() {
                return bestDist;
            }

            __device__
            inline void
            insert(int index, DistanceType dist) {
                if (dist <= bestDist) {
                    bestIndex = index;
                    bestDist = dist;
                }
            }

            DistanceType *resultDist;
            int *resultIndex;

            __device__
            inline void
            setResultLocation(DistanceType *dists, int *index, int thread, int stride) {
                resultDist = dists + thread * stride;
                resultIndex = index + thread * stride;
                if (stride != 1) {
                    for (int i = 1; i < stride; i++) {
                        resultDist[i] = INFINITY;
                        resultIndex[i] = -1;
                    }
                }
            }

            __device__
            inline void
            finish() {
                resultDist[0] = bestDist;
                resultIndex[0] = bestIndex;
            }
        };

        template<typename DistanceType, bool useHeap>
        struct KnnResultSet {
            int foundNeighbors;
            DistanceType largestHeapDist;
            int maxDistIndex;
            const int k;
            const bool sorted;
            const DistanceType epsError;


            __device__ __host__
            KnnResultSet(int knn, bool sortResults, DistanceType eps) : foundNeighbors(0), largestHeapDist(INFINITY),
                                                                        k(knn), sorted(sortResults), epsError(eps) {}

            __device__
            inline DistanceType
            worstDist() {
                return largestHeapDist;
            }

            __device__
            inline void
            insert(int index, DistanceType dist) {
                if (foundNeighbors < k) {
                    resultDist[foundNeighbors] = dist;
                    resultIndex[foundNeighbors] = index;
                    if (foundNeighbors == k - 1) {
                        if (useHeap) {
                            make_heap(resultDist, resultIndex, k, GreaterThan<DistanceType>());
                            largestHeapDist = resultDist[0];
                        } else {
                            findLargestDistIndex();
                        }

                    }
                    foundNeighbors++;
                } else if (dist < largestHeapDist) {
                    if (useHeap) {
                        resultDist[0] = dist;
                        resultIndex[0] = index;
                        sift_down(resultDist, resultIndex, 0, k, GreaterThan<DistanceType>());
                        largestHeapDist = resultDist[0];
                    } else {
                        resultDist[maxDistIndex] = dist;
                        resultIndex[maxDistIndex] = index;
                        findLargestDistIndex();
                    }

                }
            }

            __device__
            void
            findLargestDistIndex() {
                largestHeapDist = resultDist[0];
                maxDistIndex = 0;
                for (int i = 1; i < k; i++)
                    if (resultDist[i] > largestHeapDist) {
                        maxDistIndex = i;
                        largestHeapDist = resultDist[i];
                    }
            }

            float *resultDist;
            int *resultIndex;

            __device__ inline void setResultLocation(DistanceType *dists, int *index, int thread, int stride) {
                resultDist = dists + stride * thread;
                resultIndex = index + stride * thread;
                for (int i = 0; i < stride; i++) {
                    resultDist[i] = INFINITY;
                    resultIndex[i] = -1;
                }
            }

            __host__ __device__ inline void finish() {
                if (sorted) {
                    if (!useHeap) make_heap(resultDist, resultIndex, k, GreaterThan<DistanceType>());
                    for (int i = k - 1; i > 0; i--) {
                        cuda::swap(resultDist[0], resultDist[i]);
                        cuda::swap(resultIndex[0], resultIndex[i]);
                        sift_down(resultDist, resultIndex, 0, i, GreaterThan<DistanceType>());
                    }
                }
            }
        };

        template<typename DistanceType>
        struct CountingRadiusResultSet {
            int count_;
            DistanceType radius_sq_;
            int max_neighbors_;

            __device__ __host__
            CountingRadiusResultSet(DistanceType radius, int max_neighbors) : count_(0), radius_sq_(radius),
                                                                              max_neighbors_(max_neighbors) {}

            __device__
            inline DistanceType
            worstDist() {
                return radius_sq_;
            }

            __device__
            inline void
            insert(int index, float dist) {
                if (dist < radius_sq_) {
                    count_++;
                }
            }

            int *resultIndex;

            __device__
            inline void
            setResultLocation(DistanceType * /*dists*/, int *count, int thread, int stride) {
                resultIndex = count + thread * stride;
            }

            __device__
            inline void
            finish() {
                if ((max_neighbors_ <= 0) || (count_ <= max_neighbors_)) resultIndex[0] = count_;
                else resultIndex[0] = max_neighbors_;
            }
        };

// Difference to RadiusKnnResultSet: Works like KnnResultSet, doesn't pack the results densely (as the RadiusResultSet does)
        template<typename DistanceType, bool useHeap>
        struct KnnRadiusResultSet {
            int foundNeighbors;
            DistanceType largestHeapDist;
            int maxDistIndex;
            const int k;
            const bool sorted;
            const DistanceType epsError;
            const DistanceType radius_sq;


            __device__ __host__
            KnnRadiusResultSet(int knn, bool sortResults, DistanceType eps, DistanceType radius) : foundNeighbors(0),
                                                                                                   largestHeapDist(
                                                                                                           radius),
                                                                                                   k(knn),
                                                                                                   sorted(sortResults),
                                                                                                   epsError(eps),
                                                                                                   radius_sq(radius) {}

            __device__
            inline DistanceType
            worstDist() {
                return largestHeapDist;
            }

            __device__
            inline void
            insert(int index, DistanceType dist) {
                if (dist < largestHeapDist) {
                    if (foundNeighbors < k) {
                        resultDist[foundNeighbors] = dist;
                        resultIndex[foundNeighbors] = index;
                        if (foundNeighbors == k - 1) {
                            if (useHeap) {
                                make_heap(resultDist, resultIndex, k, GreaterThan<DistanceType>());
                                largestHeapDist = resultDist[0];
                            } else {
                                findLargestDistIndex();
                            }
                        }
                        foundNeighbors++;
                    } else { //if( dist < largestHeapDist )
                        if (useHeap) {
                            resultDist[0] = dist;
                            resultIndex[0] = index;
                            sift_down(resultDist, resultIndex, 0, k, GreaterThan<DistanceType>());
                            largestHeapDist = resultDist[0];
                        } else {
                            resultDist[maxDistIndex] = dist;
                            resultIndex[maxDistIndex] = index;
                            findLargestDistIndex();
                        }
                    }
                }
            }

            __device__
            void
            findLargestDistIndex() {
                largestHeapDist = resultDist[0];
                maxDistIndex = 0;
                for (int i = 1; i < k; i++)
                    if (resultDist[i] > largestHeapDist) {
                        maxDistIndex = i;
                        largestHeapDist = resultDist[i];
                    }
            }

            DistanceType *resultDist;
            int *resultIndex;

            __device__ inline void setResultLocation(DistanceType *dists, int *index, int thread, int stride) {
                resultDist = dists + stride * thread;
                resultIndex = index + stride * thread;
                for (int i = 0; i < stride; i++) {
                    resultDist[i] = INFINITY;
                    resultIndex[i] = -1;
                }
            }

            __device__ inline void finish() {
                if (sorted) {
                    if (!useHeap) make_heap(resultDist, resultIndex, k, GreaterThan<DistanceType>());
                    for (int i = k - 1; i > 0; i--) {
                        cuda::swap(resultDist[0], resultDist[i]);
                        cuda::swap(resultIndex[0], resultIndex[i]);
                        sift_down(resultDist, resultIndex, 0, i, GreaterThan<DistanceType>());
                    }
                }
            }
        };
    }
}


namespace tinyflann {

    namespace cuda {
        namespace kd_tree_builder_detail {
//! normal node: contains the split dimension and value
//! leaf node: left == index of first points, right==index of last point +1
            struct SplitInfo {
                union {
                    struct {
                        // begin of child nodes
                        int left;
                        // end of child nodes
                        int right;
                    };
                    struct {
                        int split_dim;
                        float split_val;
                    };
                };

            };

            struct IsEven {
                typedef int result_type;

                __device__
                int operator()(int i) {
                    return (i & 1) == 0;
                }
            };

            struct SecondElementIsEven {
                __host__ __device__
                bool operator()(const thrust::tuple<int, int> &i) {
                    return (thrust::get<1>(i) & 1) == 0;
                }
            };

//! just for convenience: access a float4 by an index in [0,1,2]
//! (casting it to a float* and accessing it by the index is way slower...)
            __host__ __device__
            float get_value_by_index(const float4 &f, int i) {
                switch (i) {
                    case 0:
                        return f.x;
                    case 1:
                        return f.y;
                    default:
                        return f.z;
                }

            }

//! mark a point as belonging to the left or right child of its current parent
//! called after parents are split
            struct MovePointsToChildNodes {
                MovePointsToChildNodes(int *child1, SplitInfo *splits, float *x, float *y, float *z, int *ox, int *oy,
                                       int *oz, int *lrx, int *lry, int *lrz)
                        : child1_(child1), splits_(splits), x_(x), y_(y), z_(z), ox_(ox), oy_(oy), oz_(oz), lrx_(lrx),
                          lry_(lry), lrz_(lrz) {}

                //  int dim;
                //  float threshold;
                int *child1_;
                SplitInfo *splits_;

                // coordinate values
                float *x_, *y_, *z_;
                // owner indices -> which node does the point belong to?
                int *ox_, *oy_, *oz_;
                // temp info: will be set to 1 of a point is moved to the right child node, 0 otherwise
                // (used later in the scan op to separate the points of the children into continuous ranges)
                int *lrx_, *lry_, *lrz_;

                __device__
                void operator()(const thrust::tuple<int, int, int, int> &data) {
                    int index = thrust::get<0>(data);
                    int owner = ox_[index]; // before a split, all points at the same position in the index array have the same owner
                    int point_ind1 = thrust::get<1>(data);
                    int point_ind2 = thrust::get<2>(data);
                    int point_ind3 = thrust::get<3>(data);
                    int leftChild = child1_[owner];
                    int split_dim;
                    float dim_val1, dim_val2, dim_val3;
                    SplitInfo split;
                    lrx_[index] = 0;
                    lry_[index] = 0;
                    lrz_[index] = 0;
                    // this element already belongs to a leaf node -> everything alright, no need to change anything
                    if (leftChild == -1) {
                        return;
                    }
                    // otherwise: load split data, and assign this index to the new owner
                    split = splits_[owner];
                    split_dim = split.split_dim;
                    switch (split_dim) {
                        case 0:
                            dim_val1 = x_[point_ind1];
                            dim_val2 = x_[point_ind2];
                            dim_val3 = x_[point_ind3];
                            break;
                        case 1:
                            dim_val1 = y_[point_ind1];
                            dim_val2 = y_[point_ind2];
                            dim_val3 = y_[point_ind3];
                            break;
                        default:
                            dim_val1 = z_[point_ind1];
                            dim_val2 = z_[point_ind2];
                            dim_val3 = z_[point_ind3];
                            break;

                    }


                    int r1 = leftChild + (dim_val1 > split.split_val);
                    ox_[index] = r1;
                    int r2 = leftChild + (dim_val2 > split.split_val);
                    oy_[index] = r2;
                    oz_[index] = leftChild + (dim_val3 > split.split_val);

                    lrx_[index] = (dim_val1 > split.split_val);
                    lry_[index] = (dim_val2 > split.split_val);
                    lrz_[index] = (dim_val3 > split.split_val);
                    //                      return thrust::make_tuple( r1, r2, leftChild+(dim_val > split.split_val) );
                }
            };

//! used to update the left/right pointers and aabb infos after the node splits
            struct SetLeftAndRightAndAABB {
                int maxPoints;
                int nElements;

                SplitInfo *nodes;
                int *counts;
                int *labels;
                float4 *aabbMin;
                float4 *aabbMax;
                const float *x, *y, *z;
                const int *ix, *iy, *iz;

                __host__ __device__
                void operator()(int i) {
                    int index = labels[i];
                    int right;
                    int left = counts[i];
                    nodes[index].left = left;
                    if (i < nElements - 1) {
                        right = counts[i + 1];
                    } else { // index==nNodes
                        right = maxPoints;
                    }
                    nodes[index].right = right;
                    aabbMin[index].x = x[ix[left]];
                    aabbMin[index].y = y[iy[left]];
                    aabbMin[index].z = z[iz[left]];
                    aabbMax[index].x = x[ix[right - 1]];
                    aabbMax[index].y = y[iy[right - 1]];
                    aabbMax[index].z = z[iz[right - 1]];
                }
            };


//! - decide whether a node has to be split
//! if yes:
//! - allocate child nodes
//! - set split axis as axis of maximum aabb length
            struct SplitNodes {
                int maxPointsPerNode;
                int *node_count;
                int *nodes_allocated;
                int *out_of_space;
                int *child1_;
                int *parent_;
                SplitInfo *splits;

                __device__
                void operator()(
                        thrust::tuple<int &, int &, SplitInfo &, float4 &, float4 &, int> node) // float4: aabbMin, aabbMax
                {
                    int &parent = thrust::get<0>(node);
                    int &child1 = thrust::get<1>(node);
                    SplitInfo &s = thrust::get<2>(node);
                    const float4 &aabbMin = thrust::get<3>(node);
                    const float4 &aabbMax = thrust::get<4>(node);
                    int my_index = thrust::get<5>(node);
                    bool split_node = false;
                    // first, each thread block counts the number of nodes that it needs to allocate...
                    __shared__ int block_nodes_to_allocate;
                    if (threadIdx.x == 0) block_nodes_to_allocate = 0;
                    __syncthreads();

                    // don't split if all points are equal
                    // (could lead to an infinite loop, and doesn't make any sense anyway)
                    bool all_points_in_node_are_equal =
                            aabbMin.x == aabbMax.x && aabbMin.y == aabbMax.y && aabbMin.z == aabbMax.z;

                    int offset_to_global = 0;

                    // maybe this could be replaced with a reduction...
                    if ((child1 == -1) && (s.right - s.left > maxPointsPerNode) &&
                        !all_points_in_node_are_equal) { // leaf node
                        split_node = true;
                        offset_to_global = atomicAdd(&block_nodes_to_allocate, 2);
                    }

                    __syncthreads();
                    __shared__ int block_left;
                    __shared__ bool enough_space;
                    // ... then the first thread tries to allocate this many nodes...
                    if (threadIdx.x == 0) {
                        block_left = atomicAdd(node_count, block_nodes_to_allocate);
                        enough_space = block_left + block_nodes_to_allocate < *nodes_allocated;
                        // if it doesn't succeed, no nodes will be created by this block
                        if (!enough_space) {
                            atomicAdd(node_count, -block_nodes_to_allocate);
                            *out_of_space = 1;
                        }
                    }

                    __syncthreads();
                    // this thread needs to split it's node && there was enough space for all the nodes
                    // in this block.
                    //(The whole "allocate-per-block-thing" is much faster than letting each element allocate
                    // its space on its own, because shared memory atomics are A LOT faster than
                    // global mem atomics!)
                    if (split_node && enough_space) {
                        int left = block_left + offset_to_global;

                        splits[left].left = s.left;
                        splits[left].right = s.right;
                        splits[left + 1].left = 0;
                        splits[left + 1].right = 0;

                        // split axis/position: middle of longest aabb extent
                        float4 aabbDim = aabbMax - aabbMin;
                        int maxDim = 0;
                        float maxDimLength = aabbDim.x;
                        float4 splitVal = (aabbMax + aabbMin);
                        splitVal *= 0.5f;
                        for (int i = 1; i <= 2; i++) {
                            float val = get_value_by_index(aabbDim, i);
                            if (val > maxDimLength) {
                                maxDim = i;
                                maxDimLength = val;
                            }
                        }
                        s.split_dim = maxDim;
                        s.split_val = get_value_by_index(splitVal, maxDim);

                        child1_[my_index] = left;
                        splits[my_index] = s;

                        parent_[left] = my_index;
                        parent_[left + 1] = my_index;
                        child1_[left] = -1;
                        child1_[left + 1] = -1;
                    }
                }
            };


//! computes the scatter target address for the split operation, see Sengupta,Harris,Zhang,Owen: Scan Primitives for GPU Computing
//! in my use case, this is about 2x as fast as thrust::partition
            struct set_addr3 {
                const int *val_, *f_;

                int npoints_;

                __device__
                int operator()(int id) {
                    int nf = f_[npoints_ - 1] + (val_[npoints_ - 1]);
                    int f = f_[id];
                    int t = id - f + nf;
                    return val_[id] ? f : t;
                }
            };

//! converts a float4 point (xyz) to a tuple of three float vals (used to separate the
//! float4 input buffer into three arrays in the beginning of the tree build)
            struct pointxyz_to_px_py_pz {
                __device__
                thrust::tuple<float, float, float> operator()(const float4 &val) {
                    return thrust::make_tuple(val.x, val.y, val.z);
                }
            };
        } // namespace kd_tree_builder_detail

    } // namespace cuda


    class CudaKdTreeBuilder {
    public:
        CudaKdTreeBuilder(const thrust::device_vector<float4> &points, int max_leaf_size)
                : /*out_of_space_(1,0),node_count_(1,1),*/ max_leaf_size_(max_leaf_size) {
            points_ = &points;
            int prealloc = points.size() / max_leaf_size_ * 16;
            allocation_info_.resize(3);
            allocation_info_[NodeCount] = 1;
            allocation_info_[NodesAllocated] = prealloc;
            allocation_info_[OutOfSpace] = 0;

            child1_ = new thrust::device_vector<int>(prealloc, -1);
            parent_ = new thrust::device_vector<int>(prealloc, -1);
            cuda::kd_tree_builder_detail::SplitInfo s;
            s.left = 0;
            s.right = 0;
            splits_ = new thrust::device_vector<cuda::kd_tree_builder_detail::SplitInfo>(prealloc, s);
            s.right = points.size();
            (*splits_)[0] = s;

            aabb_min_ = new thrust::device_vector<float4>(prealloc);
            aabb_max_ = new thrust::device_vector<float4>(prealloc);

            index_x_ = new thrust::device_vector<int>(points_->size());
            index_y_ = new thrust::device_vector<int>(points_->size());
            index_z_ = new thrust::device_vector<int>(points_->size());

            owners_x_ = new thrust::device_vector<int>(points_->size(), 0);
            owners_y_ = new thrust::device_vector<int>(points_->size(), 0);
            owners_z_ = new thrust::device_vector<int>(points_->size(), 0);

            leftright_x_ = new thrust::device_vector<int>(points_->size(), 0);
            leftright_y_ = new thrust::device_vector<int>(points_->size(), 0);
            leftright_z_ = new thrust::device_vector<int>(points_->size(), 0);

            tmp_index_ = new thrust::device_vector<int>(points_->size());
            tmp_owners_ = new thrust::device_vector<int>(points_->size());
            tmp_misc_ = new thrust::device_vector<int>(points_->size());

            points_x_ = new thrust::device_vector<float>(points_->size());
            points_y_ = new thrust::device_vector<float>(points_->size());
            points_z_ = new thrust::device_vector<float>(points_->size());
            delete_node_info_ = false;
        }

        ~CudaKdTreeBuilder() {
            if (delete_node_info_) {
                delete child1_;
                delete parent_;
                delete splits_;
                delete aabb_min_;
                delete aabb_max_;
                delete index_x_;
            }

            delete index_y_;
            delete index_z_;
            delete owners_x_;
            delete owners_y_;
            delete owners_z_;
            delete points_x_;
            delete points_y_;
            delete points_z_;
            delete leftright_x_;
            delete leftright_y_;
            delete leftright_z_;
            delete tmp_index_;
            delete tmp_owners_;
            delete tmp_misc_;
        }

        //! build the tree
        //! general idea:
        //! - build sorted lists of the points in x y and z order (to be able to compute tight AABBs in O(1) )
        //! - while( nodes to split exist )
        //!    - split non-child nodes along longest axis if the number of points is > max_points_per_node
        //!    - for each point: determine whether it is in a node that was split. If yes, mark it as belonging to the left or right child node of its current parent node
        //!    - reorder the points so that the points of a single node are continuous in the node array
        //!    - update the left/right pointers and AABBs of all nodes
        void buildTree() {
            //              std::cout<<"buildTree()"<<std::endl;
            //              sleep(1);
            //              Util::Timer stepTimer;
            thrust::transform(points_->begin(), points_->end(), thrust::make_zip_iterator(
                    thrust::make_tuple(points_x_->begin(), points_y_->begin(), points_z_->begin())),
                              cuda::kd_tree_builder_detail::pointxyz_to_px_py_pz());

            thrust::counting_iterator<int> it(0);
            thrust::copy(it, it + points_->size(), index_x_->begin());

            thrust::copy(index_x_->begin(), index_x_->end(), index_y_->begin());
            thrust::copy(index_x_->begin(), index_x_->end(), index_z_->begin());

            thrust::device_vector<float> tmpv(points_->size());

            // create sorted index list -> can be used to compute AABBs in O(1)
            thrust::copy(points_x_->begin(), points_x_->end(), tmpv.begin());
            thrust::sort_by_key(tmpv.begin(), tmpv.end(), index_x_->begin());
            thrust::copy(points_y_->begin(), points_y_->end(), tmpv.begin());
            thrust::sort_by_key(tmpv.begin(), tmpv.end(), index_y_->begin());
            thrust::copy(points_z_->begin(), points_z_->end(), tmpv.begin());
            thrust::sort_by_key(tmpv.begin(), tmpv.end(), index_z_->begin());


            (*aabb_min_)[0] = make_float4((*points_x_)[(*index_x_)[0]], (*points_y_)[(*index_y_)[0]],
                                          (*points_z_)[(*index_z_)[0]], 0);

            (*aabb_max_)[0] = make_float4((*points_x_)[(*index_x_)[points_->size() - 1]],
                                          (*points_y_)[(*index_y_)[points_->size() - 1]],
                                          (*points_z_)[(*index_z_)[points_->size() - 1]], 0);
#ifdef PRINT_DEBUG_TIMING
            hipDeviceSynchronize();
        std::cout<<" initial stuff:"<<stepTimer.elapsed()<<std::endl;
        stepTimer.restart();
#endif
            int last_node_count = 0;
            for (int i = 0;; i++) {
                cuda::kd_tree_builder_detail::SplitNodes sn;

                sn.maxPointsPerNode = max_leaf_size_;
                sn.node_count = thrust::raw_pointer_cast(&allocation_info_[NodeCount]);
                sn.nodes_allocated = thrust::raw_pointer_cast(&allocation_info_[NodesAllocated]);
                sn.out_of_space = thrust::raw_pointer_cast(&allocation_info_[OutOfSpace]);
                sn.child1_ = thrust::raw_pointer_cast(&(*child1_)[0]);
                sn.parent_ = thrust::raw_pointer_cast(&(*parent_)[0]);
                sn.splits = thrust::raw_pointer_cast(&(*splits_)[0]);

                thrust::counting_iterator<int> cit(0);
                thrust::for_each(thrust::make_zip_iterator(
                        thrust::make_tuple(parent_->begin(), child1_->begin(), splits_->begin(), aabb_min_->begin(),
                                           aabb_max_->begin(), cit)),
                                 thrust::make_zip_iterator(thrust::make_tuple(parent_->begin() + last_node_count,
                                                                              child1_->begin() + last_node_count,
                                                                              splits_->begin() + last_node_count,
                                                                              aabb_min_->begin() + last_node_count,
                                                                              aabb_max_->begin() + last_node_count,
                                                                              cit + last_node_count)),
                                 sn);
                // copy allocation info to host
                thrust::host_vector<int> alloc_info = allocation_info_;

                if (last_node_count == alloc_info[NodeCount]) { // no more nodes were split -> done
                    break;
                }
                last_node_count = alloc_info[NodeCount];

                // a node was un-splittable due to a lack of space
                if (alloc_info[OutOfSpace] == 1) {
                    resize_node_vectors(alloc_info[NodesAllocated] * 2);
                    alloc_info[OutOfSpace] = 0;
                    alloc_info[NodesAllocated] *= 2;
                    allocation_info_ = alloc_info;
                }
#ifdef PRINT_DEBUG_TIMING
                hipDeviceSynchronize();
            std::cout<<" node split:"<<stepTimer.elapsed()<<std::endl;
            stepTimer.restart();
#endif

                // foreach point: point was in node that was split?move it to child (leaf) node : do nothing
                cuda::kd_tree_builder_detail::MovePointsToChildNodes sno(thrust::raw_pointer_cast(&(*child1_)[0]),
                                                                         thrust::raw_pointer_cast(&(*splits_)[0]),
                                                                         thrust::raw_pointer_cast(&(*points_x_)[0]),
                                                                         thrust::raw_pointer_cast(&(*points_y_)[0]),
                                                                         thrust::raw_pointer_cast(&(*points_z_)[0]),
                                                                         thrust::raw_pointer_cast(&(*owners_x_)[0]),
                                                                         thrust::raw_pointer_cast(&(*owners_y_)[0]),
                                                                         thrust::raw_pointer_cast(&(*owners_z_)[0]),
                                                                         thrust::raw_pointer_cast(&(*leftright_x_)[0]),
                                                                         thrust::raw_pointer_cast(&(*leftright_y_)[0]),
                                                                         thrust::raw_pointer_cast(&(*leftright_z_)[0])
                );
                thrust::counting_iterator<int> ci0(0);
                thrust::for_each(thrust::make_zip_iterator(
                        thrust::make_tuple(ci0, index_x_->begin(), index_y_->begin(), index_z_->begin())),
                                 thrust::make_zip_iterator(
                                         thrust::make_tuple(ci0 + points_->size(), index_x_->end(), index_y_->end(),
                                                            index_z_->end())), sno);

#ifdef PRINT_DEBUG_TIMING
                hipDeviceSynchronize();
            std::cout<<" set new owners:"<<stepTimer.elapsed()<<std::endl;
            stepTimer.restart();
#endif

                // move points around so that each leaf node's points are continuous
                separate_left_and_right_children(*index_x_, *owners_x_, *tmp_index_, *tmp_owners_, *leftright_x_);
                std::swap(tmp_index_, index_x_);
                std::swap(tmp_owners_, owners_x_);
                separate_left_and_right_children(*index_y_, *owners_y_, *tmp_index_, *tmp_owners_, *leftright_y_,
                                                 false);
                std::swap(tmp_index_, index_y_);
                separate_left_and_right_children(*index_z_, *owners_z_, *tmp_index_, *tmp_owners_, *leftright_z_,
                                                 false);
                std::swap(tmp_index_, index_z_);

#ifdef PRINT_DEBUG_TIMING
                hipDeviceSynchronize();
            std::cout<<" split:"<<stepTimer.elapsed()<<std::endl;
            stepTimer.restart();
#endif
                // calculate new AABB etc
                update_leftright_and_aabb(*points_x_, *points_y_, *points_z_, *index_x_, *index_y_, *index_z_,
                                          *owners_x_, *splits_, *aabb_min_, *aabb_max_);
#ifdef PRINT_DEBUG_TIMING
                hipDeviceSynchronize();
            std::cout<<" update_leftright_and_aabb:"<<stepTimer.elapsed()<<std::endl;
            stepTimer.restart();
            print_vector(node_count_);
#endif

            }
        }

        template<class Distance>
        friend
        class KDTreeCuda3dIndex;

    protected:


        //! takes the partitioned nodes, and sets the left-/right info of leaf nodes, as well as the AABBs
        void
        update_leftright_and_aabb(const thrust::device_vector<float> &x, const thrust::device_vector<float> &y,
                                  const thrust::device_vector<float> &z,
                                  const thrust::device_vector<int> &ix, const thrust::device_vector<int> &iy,
                                  const thrust::device_vector<int> &iz,
                                  const thrust::device_vector<int> &owners,
                                  thrust::device_vector<cuda::kd_tree_builder_detail::SplitInfo> &splits,
                                  thrust::device_vector<float4> &aabbMin, thrust::device_vector<float4> &aabbMax) {
            thrust::device_vector<int> *labelsUnique = tmp_owners_;
            thrust::device_vector<int> *countsUnique = tmp_index_;
            // assume: points of each node are continuous in the array

            // find which nodes are here, and where each node's points begin and end
            int unique_labels =
                    thrust::unique_by_key_copy(owners.begin(), owners.end(), thrust::counting_iterator<int>(0),
                                               labelsUnique->begin(), countsUnique->begin()).first -
                    labelsUnique->begin();

            // update the info
            cuda::kd_tree_builder_detail::SetLeftAndRightAndAABB s;
            s.maxPoints = x.size();
            s.nElements = unique_labels;
            s.nodes = thrust::raw_pointer_cast(&(splits[0]));
            s.counts = thrust::raw_pointer_cast(&((*countsUnique)[0]));
            s.labels = thrust::raw_pointer_cast(&((*labelsUnique)[0]));
            s.x = thrust::raw_pointer_cast(&x[0]);
            s.y = thrust::raw_pointer_cast(&y[0]);
            s.z = thrust::raw_pointer_cast(&z[0]);
            s.ix = thrust::raw_pointer_cast(&ix[0]);
            s.iy = thrust::raw_pointer_cast(&iy[0]);
            s.iz = thrust::raw_pointer_cast(&iz[0]);
            s.aabbMin = thrust::raw_pointer_cast(&aabbMin[0]);
            s.aabbMax = thrust::raw_pointer_cast(&aabbMax[0]);

            thrust::counting_iterator<int> it(0);
            thrust::for_each(it, it + unique_labels, s);
        }

        //! Separates the left and right children of each node into continuous parts of the array.
        //! More specifically, it seperates children with even and odd node indices because nodes are always
        //! allocated in pairs -> child1==child2+1 -> child1 even and child2 odd, or vice-versa.
        //! Since the split operation is stable, this results in continuous partitions
        //! for all the single nodes.
        //! (basically the split primitive according to sengupta et al)
        //! about twice as fast as thrust::partition
        void separate_left_and_right_children(thrust::device_vector<int> &key_in, thrust::device_vector<int> &val_in,
                                              thrust::device_vector<int> &key_out, thrust::device_vector<int> &val_out,
                                              thrust::device_vector<int> &left_right_marks,
                                              bool scatter_val_out = true) {
            thrust::device_vector<int> *f_tmp = &val_out;
            thrust::device_vector<int> *addr_tmp = tmp_misc_;

            thrust::exclusive_scan( /*thrust::make_transform_iterator(*/
                    left_right_marks.begin() /*,cuda::kd_tree_builder_detail::IsEven*/
                    /*())*/, /*thrust::make_transform_iterator(*/
                    left_right_marks.end() /*,cuda::kd_tree_builder_detail::IsEven*/
                    /*())*/, f_tmp->begin());
            cuda::kd_tree_builder_detail::set_addr3 sa;
            sa.val_ = thrust::raw_pointer_cast(&left_right_marks[0]);
            sa.f_ = thrust::raw_pointer_cast(&(*f_tmp)[0]);
            sa.npoints_ = key_in.size();
            thrust::counting_iterator<int> it(0);
            thrust::transform(it, it + val_in.size(), addr_tmp->begin(), sa);

            thrust::scatter(key_in.begin(), key_in.end(), addr_tmp->begin(), key_out.begin());
            if (scatter_val_out) thrust::scatter(val_in.begin(), val_in.end(), addr_tmp->begin(), val_out.begin());
        }

        //! allocates additional space in all the node-related vectors.
        //! new_size elements will be added to all vectors.
        void resize_node_vectors(size_t new_size) {
            size_t add = new_size - child1_->size();
            child1_->insert(child1_->end(), add, -1);
            parent_->insert(parent_->end(), add, -1);
            cuda::kd_tree_builder_detail::SplitInfo s;
            s.left = 0;
            s.right = 0;
            splits_->insert(splits_->end(), add, s);
            float4 f;
            aabb_min_->insert(aabb_min_->end(), add, f);
            aabb_max_->insert(aabb_max_->end(), add, f);
        }


        const thrust::device_vector<float4> *points_;

        // tree data, those are stored per-node

        //! left child of each node. (right child==left child + 1, due to the alloc mechanism)
        //! child1_[node]==-1 if node is a leaf node
        thrust::device_vector<int> *child1_;
        //! parent node of each node
        thrust::device_vector<int> *parent_;
        //! split info (dim/value or left/right pointers)
        thrust::device_vector<cuda::kd_tree_builder_detail::SplitInfo> *splits_;
        //! min aabb value of each node
        thrust::device_vector<float4> *aabb_min_;
        //! max aabb value of each node
        thrust::device_vector<float4> *aabb_max_;

        enum AllocationInfo {
            NodeCount = 0,
            NodesAllocated = 1,
            OutOfSpace = 2
        };
        // those were put into a single vector of 3 elements so that only one mem transfer will be needed for all three of them
        //  thrust::device_vector<int> out_of_space_;
        //  thrust::device_vector<int> node_count_;
        //  thrust::device_vector<int> nodes_allocated_;
        thrust::device_vector<int> allocation_info_;

        int max_leaf_size_;

        // coordinate values of the points
        thrust::device_vector<float> *points_x_, *points_y_, *points_z_;
        // indices
        thrust::device_vector<int> *index_x_, *index_y_, *index_z_;
        // owner node
        thrust::device_vector<int> *owners_x_, *owners_y_, *owners_z_;
        // contains info about whether a point was partitioned to the left or right child after a split
        thrust::device_vector<int> *leftright_x_, *leftright_y_, *leftright_z_;
        thrust::device_vector<int> *tmp_index_, *tmp_owners_, *tmp_misc_;
        bool delete_node_info_;
    };


    namespace KdTreeCudaPrivate {
        template<typename GPUResultSet, typename Distance>
        __device__ void searchNeighbors(const cuda::kd_tree_builder_detail::SplitInfo *splits,
                             const int *child1,
                             const int *parent,
                             const float4 *aabbLow,
                             const float4 *aabbHigh, const float4 *elements, const float4 &q, GPUResultSet &result,
                             const Distance &distance = Distance()) {

            bool backtrack = false;
            int lastNode = -1;
            int current = 0;

            cuda::kd_tree_builder_detail::SplitInfo split;
            while (true) {
                if (current == -1) break;
                split = splits[current];

                float diff1;
                if (split.split_dim == 0) diff1 = q.x - split.split_val;
                else if (split.split_dim == 1) diff1 = q.y - split.split_val;
                else if (split.split_dim == 2) diff1 = q.z - split.split_val;

                // children are next to each other: leftChild+1 == rightChild
                int leftChild = child1[current];
                int bestChild = leftChild;
                int otherChild = leftChild;

                if ((diff1) < 0) {
                    otherChild++;
                } else {
                    bestChild++;
                }

                if (!backtrack) {
                    /* If this is a leaf node, then do check and return. */
                    if (leftChild == -1) {
                        for (int i = split.left; i < split.right; ++i) {
                            float dist = distance.dist(elements[i], q);
                            result.insert(i, dist);
                        }
                        backtrack = true;
                        lastNode = current;
                        current = parent[current];
                    } else { // go to closer child node
                        lastNode = current;
                        current = bestChild;
                    }
                } else { // continue moving back up the tree or visit far node?
                    // minimum possible distance between query point and a point inside the AABB
                    float mindistsq = 0;
                    float4 aabbMin = aabbLow[otherChild];
                    float4 aabbMax = aabbHigh[otherChild];

                    if (q.x < aabbMin.x) mindistsq += distance.axisDist(q.x, aabbMin.x);
                    else if (q.x > aabbMax.x) mindistsq += distance.axisDist(q.x, aabbMax.x);
                    if (q.y < aabbMin.y) mindistsq += distance.axisDist(q.y, aabbMin.y);
                    else if (q.y > aabbMax.y) mindistsq += distance.axisDist(q.y, aabbMax.y);
                    if (q.z < aabbMin.z) mindistsq += distance.axisDist(q.z, aabbMin.z);
                    else if (q.z > aabbMax.z) mindistsq += distance.axisDist(q.z, aabbMax.z);

                    //  the far node was NOT the last node (== not visited yet) AND there could be a closer point in it
                    if ((lastNode == bestChild) && (mindistsq <= result.worstDist())) {
                        lastNode = current;
                        current = otherChild;
                        backtrack = false;
                    } else {
                        lastNode = current;
                        current = parent[current];
                    }
                }

            }
        }


        template<typename GPUResultSet, typename Distance>
        __global__
        void nearestKernel(const cuda::kd_tree_builder_detail::SplitInfo *splits,
                           const int *child1,
                           const int *parent,
                           const float4 *aabbMin,
                           const float4 *aabbMax, const float4 *elements, const float *query, int stride,
                           int resultStride, int *resultIndex, float *resultDist, int querysize, GPUResultSet result,
                           Distance dist = Distance()) {
            typedef float DistanceType;
            typedef float ElementType;
            //                  typedef DistanceType float;
            size_t tid = blockDim.x * blockIdx.x + threadIdx.x;

            if (tid >= querysize) return;

            float4 q = make_float4(query[tid * stride], query[tid * stride + 1], query[tid * stride + 2], 0);

            result.setResultLocation(resultDist, resultIndex, tid, resultStride);

            searchNeighbors(splits, child1, parent, aabbMin, aabbMax, elements, q, result, dist);

            result.finish();
        }

    }

//! contains some pointers that use cuda data types and that cannot be easily
//! forward-declared.
//! basically it contains all GPU buffers
    template<typename Distance>
    struct KDTreeCuda3dIndex<Distance>::GpuHelper {
        thrust::device_vector<cuda::kd_tree_builder_detail::SplitInfo> *gpu_splits_;
        thrust::device_vector<int> *gpu_parent_;
        thrust::device_vector<int> *gpu_child1_;
        thrust::device_vector<float4> *gpu_aabb_min_;
        thrust::device_vector<float4> *gpu_aabb_max_;
        thrust::device_vector<float4> *gpu_points_;
        thrust::device_vector<int> *gpu_vind_;

        GpuHelper() : gpu_splits_(0), gpu_parent_(0), gpu_child1_(0), gpu_aabb_min_(0), gpu_aabb_max_(0),
                      gpu_points_(0), gpu_vind_(0) {
        }

        ~GpuHelper() {
            delete gpu_splits_;
            gpu_splits_ = 0;
            delete gpu_parent_;
            gpu_parent_ = 0;
            delete gpu_child1_;
            gpu_child1_ = 0;
            delete gpu_aabb_max_;
            gpu_aabb_max_ = 0;
            delete gpu_aabb_min_;
            gpu_aabb_min_ = 0;
            delete gpu_vind_;
            gpu_vind_ = 0;

            delete gpu_points_;
            gpu_points_ = 0;
        }
    };

//! thrust transform functor
//! transforms indices in the internal data set back to the original indices
    struct map_indices {
        const int *v_;

        map_indices(const int *v) : v_(v) {}

        __host__ __device__
        float operator()(const int &i) const {
            if (i >= 0) return v_[i];
            else return i;
        }
    };

//! implementation of L2 distance for the CUDA kernels
    struct CudaL2 {

        static float __host__ __device__ axisDist(float a, float b) {
            return (a - b) * (a - b);
        }

        static float __host__ __device__ dist(float4 a, float4 b) {
            float4 diff = a - b;
            return dot(diff, diff);
        }
    };

//! implementation of L1 distance for the CUDA kernels
//! NOT TESTED!
    struct CudaL1 {

        static float __host__ __device__ axisDist(float a, float b) {
            return fabs(a - b);
        }

        static float __host__ __device__ dist(float4 a, float4 b) {
            return fabs(a.x - b.x) + fabs(a.y - b.y) + (a.z - b.z) + (a.w - b.w);
        }
    };


    template<typename Distance>
    void KDTreeCuda3dIndex<Distance>::knnSearch(const float* queries, size_t n_query, int istride, int* indices,
            float* dists, size_t knn, const SearchParams& params) const {
        int ostride = knn;

        int threadsPerBlock = 128;
        int blocksPerGrid = (n_query + threadsPerBlock - 1) / threadsPerBlock;

        float epsError = 1 + params.eps;
        bool sorted = params.sorted;
        bool use_heap = params.use_heap;

        Distance distance;
        if (knn == 1) {
            KdTreeCudaPrivate::nearestKernel<<<blocksPerGrid, threadsPerBlock >> > (thrust::raw_pointer_cast(
                    &((*gpu_helper_->gpu_splits_)[0])),
                    thrust::raw_pointer_cast(&((*gpu_helper_->gpu_child1_)[0])),
                    thrust::raw_pointer_cast(&((*gpu_helper_->gpu_parent_)[0])),
                    thrust::raw_pointer_cast(&((*gpu_helper_->gpu_aabb_min_)[0])),
                    thrust::raw_pointer_cast(&((*gpu_helper_->gpu_aabb_max_)[0])),
                    thrust::raw_pointer_cast(&((*gpu_helper_->gpu_points_)[0])),
                    queries,
                    istride,
                    ostride,
                    indices,
                    dists,
                    n_query, cuda::SingleResultSet<float>(epsError), distance);
        } else {
            if (use_heap) {
                KdTreeCudaPrivate::nearestKernel<<<blocksPerGrid, threadsPerBlock >> > (thrust::raw_pointer_cast(
                        &((*gpu_helper_->gpu_splits_)[0])),
                        thrust::raw_pointer_cast(&((*gpu_helper_->gpu_child1_)[0])),
                        thrust::raw_pointer_cast(&((*gpu_helper_->gpu_parent_)[0])),
                        thrust::raw_pointer_cast(&((*gpu_helper_->gpu_aabb_min_)[0])),
                        thrust::raw_pointer_cast(&((*gpu_helper_->gpu_aabb_max_)[0])),
                        thrust::raw_pointer_cast(&((*gpu_helper_->gpu_points_)[0])),
                        queries,
                        istride,
                        ostride,
                        indices,
                        dists,
                        n_query, cuda::KnnResultSet<float, true>(knn, sorted, epsError), distance);
            } else {
                KdTreeCudaPrivate::nearestKernel<<<blocksPerGrid, threadsPerBlock >> > (thrust::raw_pointer_cast(
                        &((*gpu_helper_->gpu_splits_)[0])),
                        thrust::raw_pointer_cast(&((*gpu_helper_->gpu_child1_)[0])),
                        thrust::raw_pointer_cast(&((*gpu_helper_->gpu_parent_)[0])),
                        thrust::raw_pointer_cast(&((*gpu_helper_->gpu_aabb_min_)[0])),
                        thrust::raw_pointer_cast(&((*gpu_helper_->gpu_aabb_max_)[0])),
                        thrust::raw_pointer_cast(&((*gpu_helper_->gpu_points_)[0])),
                        queries,
                        istride,
                        ostride,
                        indices,
                        dists,
                        n_query, cuda::KnnResultSet<float, false>(knn, sorted, epsError), distance
                );
            }
        }
        thrust::device_ptr<int> id = thrust::device_pointer_cast(indices);
        thrust::transform(id, id + knn * n_query, id,
                          map_indices(thrust::raw_pointer_cast(&((*gpu_helper_->gpu_vind_))[0])));
    }


//! used in the radius search to count the total number of neighbors
    struct isNotMinusOne {
        __host__ __device__
        bool operator()(int i) {
            return i != -1;
        }
    };

    template<typename Distance>
    int KDTreeCuda3dIndex<Distance>::radiusSearch(const float* queries, size_t n_query, int n_query_stride, int* indices,
            float* dists, float radius, const SearchParams& params) const {
        int max_neighbors = params.max_neighbors;
        assert(max_neighbors > 0);

        bool sorted = params.sorted;
        float epsError = 1 + params.eps;
        bool use_heap = params.use_heap;
        int istride = n_query_stride;
        int ostride = max_neighbors;

        Distance distance;
        int threadsPerBlock = 128;
        int blocksPerGrid = (n_query + threadsPerBlock - 1) / threadsPerBlock;

        if (use_heap) {
            KdTreeCudaPrivate::nearestKernel<<<blocksPerGrid, threadsPerBlock >> > (thrust::raw_pointer_cast(
                    &((*gpu_helper_->gpu_splits_)[0])),
                    thrust::raw_pointer_cast(&((*gpu_helper_->gpu_child1_)[0])),
                    thrust::raw_pointer_cast(&((*gpu_helper_->gpu_parent_)[0])),
                    thrust::raw_pointer_cast(&((*gpu_helper_->gpu_aabb_min_)[0])),
                    thrust::raw_pointer_cast(&((*gpu_helper_->gpu_aabb_max_)[0])),
                    thrust::raw_pointer_cast(&((*gpu_helper_->gpu_points_)[0])),
                    queries,
                    istride,
                    ostride,
                    indices,
                    dists,
                    n_query, cuda::KnnRadiusResultSet<float, true>(max_neighbors, sorted, epsError,
                                                                               radius), distance);
        } else {
            KdTreeCudaPrivate::nearestKernel<<<blocksPerGrid, threadsPerBlock >> > (thrust::raw_pointer_cast(
                    &((*gpu_helper_->gpu_splits_)[0])),
                    thrust::raw_pointer_cast(&((*gpu_helper_->gpu_child1_)[0])),
                    thrust::raw_pointer_cast(&((*gpu_helper_->gpu_parent_)[0])),
                    thrust::raw_pointer_cast(&((*gpu_helper_->gpu_aabb_min_)[0])),
                    thrust::raw_pointer_cast(&((*gpu_helper_->gpu_aabb_max_)[0])),
                    thrust::raw_pointer_cast(&((*gpu_helper_->gpu_points_)[0])),
                    queries,
                    istride,
                    ostride,
                    indices,
                    dists,
                    n_query, cuda::KnnRadiusResultSet<float, false>(max_neighbors, sorted, epsError,
                                                                                radius), distance);
        }

        thrust::device_ptr<int> id = thrust::device_pointer_cast(indices);
        thrust::transform(id, id + max_neighbors * n_query, id,
                          map_indices(thrust::raw_pointer_cast(&((*gpu_helper_->gpu_vind_))[0])));

        return thrust::count_if(id, id + max_neighbors * n_query, isNotMinusOne());
    }

    template<typename Distance>
    void KDTreeCuda3dIndex<Distance>::uploadTreeToGpu() {
        // just make sure that no weird alignment stuff is going on...
        // shouldn't, but who knows
        // (I would make this a (boost) static assertion, but so far flann seems to avoid boost
        //  assert( sizeof( KdTreeCudaPrivate::GpuNode)==sizeof( Node ) );
        delete gpu_helper_;
        gpu_helper_ = new GpuHelper;
        gpu_helper_->gpu_points_ = new thrust::device_vector<float4>(size_);
        thrust::device_vector<float4> tmp(size_);

        thrust::copy(thrust::device_pointer_cast((float4 *) dataset_),
                     thrust::device_pointer_cast((float4 *) (dataset_)) + size_, tmp.begin());

        CudaKdTreeBuilder builder(tmp, leaf_max_size_);
        builder.buildTree();

        gpu_helper_->gpu_splits_ = builder.splits_;
        gpu_helper_->gpu_aabb_min_ = builder.aabb_min_;
        gpu_helper_->gpu_aabb_max_ = builder.aabb_max_;
        gpu_helper_->gpu_child1_ = builder.child1_;
        gpu_helper_->gpu_parent_ = builder.parent_;
        gpu_helper_->gpu_vind_ = builder.index_x_;
        thrust::gather(builder.index_x_->begin(), builder.index_x_->end(), tmp.begin(), gpu_helper_->gpu_points_->begin());
    }


    template<typename Distance>
    void KDTreeCuda3dIndex<Distance>::clearGpuBuffers() {
        delete gpu_helper_;
        gpu_helper_ = 0;
    }

    // instantiations for L1 and L2.
    template void KDTreeCuda3dIndex<CudaL1>::uploadTreeToGpu();
    template void KDTreeCuda3dIndex<CudaL2>::uploadTreeToGpu();

    template void KDTreeCuda3dIndex<CudaL1>::clearGpuBuffers();
    template void KDTreeCuda3dIndex<CudaL2>::clearGpuBuffers();

    template struct KDTreeCuda3dIndex<CudaL1 >::GpuHelper;
    template struct KDTreeCuda3dIndex<CudaL2 >::GpuHelper;

    template void KDTreeCuda3dIndex<CudaL1>::knnSearch(const float* queries, size_t n_query, int n_query_stride, int* indices, float* dists, size_t knn, const SearchParams& params) const;
    template void KDTreeCuda3dIndex<CudaL2>::knnSearch(const float* queries, size_t n_query, int n_query_stride, int* indices, float* dists, size_t knn, const SearchParams& params) const;

    template int KDTreeCuda3dIndex<CudaL1>::radiusSearch(const float* queries, size_t n_query, int n_query_stride, int* indices, float* dists, float radius, const SearchParams& params) const;
    template int KDTreeCuda3dIndex<CudaL2>::radiusSearch(const float* queries, size_t n_query, int n_query_stride, int* indices, float* dists, float radius, const SearchParams& params) const;

};
